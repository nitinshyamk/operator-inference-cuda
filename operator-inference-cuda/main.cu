
#include "hip/hip_runtime.h"
#include ""

#include "cuda_host_matrix.h"
#include "cuda_gpu_matrix.h"
#include "cuda_libraries.h"
#include "ddt.cuh"
#include "gpu_timer.h"
#include "linear_algebra.cuh"

#include <stdio.h>
#include <iostream>

int main()
{
    gpu_timer<true> block_timer("int main()");
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    using std::cout;
    using std::endl;

    cuda_libraries cuLibraries;
    linear_algebra linalg(cuLibraries);
    Ddt<ForwardDifference4> ddt(0.1);

    gpu_timer<false> timer("main timer");

    timer.start();
    cuda_host_matrix A(10, 10, cuda_host_matrix::MatrixType::CM_DENSE);
    for (int r = 0; r < 10 * 10; ++r)
        A.data.get()[r] = r/ 10.0;

    cuda_gpu_matrix Agpu(10, 10);
    A.copyToGpuMemory(Agpu);

    timer.stop();
    timer.elapsed();

    cuda_gpu_matrix ddt_ans = ddt(Agpu);

    timer.start();
    auto Cgpu = linalg.multiply(Agpu, false, Agpu, false);
    cuda_host_matrix C(Cgpu.M(), Cgpu.N(), cuda_host_matrix::MatrixType::CM_DENSE);
    timer.stop();
    timer.elapsed();

    C.copyFromGpuMemory(Cgpu);

    A.print();
    C.print();
    return 0;
}
