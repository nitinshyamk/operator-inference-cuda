#include "hip/hip_runtime.h"
#include "LinearAlgebra.cuh"

cuda_gpu_matrix
linear_algebra::multiply(
	const cuda_gpu_matrix& A,
	bool transposeA,
	const cuda_gpu_matrix& B,
	bool transposeB) const
{
	int A_M = transposeA ? A.N() : A.M();
	int A_N = transposeA ? A.M() : A.N();
	int B_M = transposeB ? B.N() : B.M();
	int B_N = transposeB ? B.M() : B.N();

	if (A_N != B_M)
		throw incompatible_dimensions_error(A_M, A_N, B_M, B_N);

	cuda_gpu_matrix C(A_M, B_N);

	hipblasLtMatmulDesc_t op_description = NULL;
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatmulDescCreate(&op_description, HIPBLAS_COMPUTE_64F, HIP_R_64F));
	hipblasOperation_t shouldTransposeValue = HIPBLAS_OP_T;

	auto setTransposeAttribute = [&op_description, &shouldTransposeValue](hipblasLtMatmulDescAttributes_t attr) -> void
	{
		checkCudaStatus<cublas_matrix_operation_error>(
			hipblasLtMatmulDescSetAttribute(
				op_description,
				attr,
				&shouldTransposeValue,
				sizeof(shouldTransposeValue)));
	};

	if (transposeA)
		setTransposeAttribute(HIPBLASLT_MATMUL_DESC_TRANSA);

	if (transposeB)
		setTransposeAttribute(HIPBLASLT_MATMUL_DESC_TRANSB);

	// create descriptions
	hipblasLtMatrixLayout_t Adescription, Bdescription, Cdescription;
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatrixLayoutCreate(&Adescription, HIP_R_64F, A_M, A_N, A_M));
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatrixLayoutCreate(&Bdescription, HIP_R_64F, B_M, B_N, B_M));
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatrixLayoutCreate(&Cdescription, HIP_R_64F, C.M(), C.N(), C.M()));

	size_t workspace_size = 1 << 16;
	std::shared_ptr<void> workspace = allocate_on_device<void>(workspace_size);

	hipblasLtMatmulPreference_t preference = NULL;
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatmulPreferenceCreate(&preference));
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasLtMatmulPreferenceSetAttribute(
			preference,
			HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
			&workspace_size,
			sizeof(workspace_size)));

	hipblasLtMatmulHeuristicResult_t heuristicResult = {};
	int returnedResults = 0;
	hipblasLtMatmulAlgoGetHeuristic(
		cudalibraries.get_blaslt_handle(),
		op_description,
		Adescription,
		Bdescription,
		Cdescription,
		Cdescription,
		preference,
		1,
		&heuristicResult,
		&returnedResults);

	double alpha = 1, beta = 0;
	hipblasLtMatmul(
		cudalibraries.get_blaslt_handle(),
		op_description,
		&alpha,
		A.c_ptr(),
		Adescription,
		B.c_ptr(),
		Bdescription,
		&beta,
		// currently C must equal D, so we just pass it in twice
		C.c_ptr(),
		Cdescription,
		C.c_ptr(),
		Cdescription,
		&heuristicResult.algo,
		workspace.get(),
		workspace_size,
		0); // default cudastream

	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return C;
};

cuda_gpu_vector
linear_algebra::add(
	const cuda_gpu_vector& a,
	double alpha,
	const cuda_gpu_vector& b,
	double beta)
{
	if (a.M() != b.M())
		throw incompatible_dimensions_error(a.M(), a.N(), b.M(), b.N());

	cuda_gpu_vector ans(a.M());
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasDaxpy(cudalibraries.get_blas_handle(), a.M(), &alpha, a.c_ptr(), 1, ans.c_ptr(), 1));

	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasDaxpy(cudalibraries.get_blas_handle(), a.M(), &beta, b.c_ptr(), 1, ans.c_ptr(), 1));

	return ans;
}


cuda_gpu_matrix
linear_algebra::concatenate(
		const cuda_gpu_matrix& A,
		const cuda_gpu_matrix& B,
		bool shouldConcatenateVertically) const
{
	if (shouldConcatenateVertically && A.N() != B.N() || !shouldConcatenateVertically && A.M() != B.M())
		throw std::invalid_argument("Invalid dimensions for specified concatenation operation");
	
	double* Asrc = A.c_ptr();
	double* Bsrc = B.c_ptr();
	if (!shouldConcatenateVertically)
	{
		cuda_gpu_matrix ans(A.M(), A.N() + B.N());
		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr(),
			Asrc,
			A.M() * A.N() * sizeof((*Asrc)),
			hipMemcpyDeviceToDevice));

		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + A.M() * A.N(),
			Bsrc,
			B.M() * B.N() * sizeof((*Bsrc)),
			hipMemcpyDeviceToDevice));
		return ans;
	}

	cuda_gpu_matrix ans(A.M() + B.M(), A.N());
	for (size_t col = 0; col < A.N(); ++col)
	{
		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + columnMajorZeroIndex(0, col, ans.M(), ans.N()),
			Asrc + columnMajorZeroIndex(0, col, A.M(), A.N()),
			A.M() * sizeof((*Asrc)),
			hipMemcpyDeviceToDevice));

		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + columnMajorZeroIndex(A.M(), col, ans.M(), ans.N()),
			Bsrc + columnMajorZeroIndex(0, col, B.M(), B.N()),
			B.M() * sizeof((*Bsrc)),
			hipMemcpyDeviceToDevice));
	}
	return ans;
}

svd
linear_algebra::svd_decomposition(const cuda_gpu_matrix& A) const
{
	int lwork;
	checkCudaStatus<cuda_memory_error>(
		hipsolverDnSgesvd_bufferSize(this->cudalibraries.get_solver_handle(), A.M(), A.N(), &lwork));

	std::shared_ptr<double> workspace = allocate_on_device<double>(sizeof(double) * lwork);

	signed char jobu = 'A', jobvt = 'A';

	size_t diag_length = A.M() < A.N() ? A.M() : A.N();
	cuda_gpu_matrix U(A.M(), A.M());
	cuda_gpu_vector Svec(diag_length);
	cuda_gpu_vector unconverged(diag_length);
	cuda_gpu_matrix Vt(A.N(), A.N());

	double* d_work, *d_rwork;
	int* dev_info = 0;

	hipsolverDnDgesvd(
		this->cudalibraries.get_solver_handle(),
		jobu,
		jobvt,
		A.M(),
		A.N(),
		A.c_ptr(),
		A.M(),
		Svec.c_ptr(),
		U.c_ptr(),
		A.M(),  // ldu
		Vt.c_ptr(),
		A.N(), // ldvt,
		workspace.get(),
		lwork,
		unconverged.c_ptr(),
		dev_info);

	cuda_gpu_matrix S(A.M(), A.N());
	for (size_t i = 0; i < diag_length; ++i)
	{
		hipMemcpy(
			S.c_ptr() + columnMajorZeroIndex(i, i, A.M(), A.N()),
			Svec.c_ptr() + columnMajorZeroIndex(i, 1, Svec.M(), Svec.N()),
			sizeof(double),
			hipMemcpyDeviceToDevice);
	}
	svd ans{ U, S, Vt };
	return ans;
}

cuda_gpu_matrix
linear_algebra::pinv(const cuda_gpu_matrix& A) const
{
	return pinv(svd_decomposition(A));
}

cuda_gpu_matrix
linear_algebra::pinv(const svd& decomposition) const
{
	cuda_gpu_matrix pinvSigma = decomposition.S.deep_copy();

	// invert diagonal matrix kernel
	size_t diag_length = pinvSigma.M() > pinvSigma.N() ? pinvSigma.N() : pinvSigma.M();

	size_t blockDim = 1 << 6;
	size_t gridDim = (diag_length + blockDim - 1) / blockDim;
	
	invert_rectangular_diagonal_kernel KERNEL_ARGS2(gridDim, blockDim) (pinvSigma.c_ptr(), pinvSigma.M(), pinvSigma.N());
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());

	cuda_gpu_matrix pinv = multiply(
		multiply(
			decomposition.Vt, true,
			pinvSigma, true), false, 
		decomposition.U, true);

	return pinv;
}

cuda_gpu_matrix
linear_algebra::subset(
	const cuda_gpu_matrix& A,
	std::pair<size_t, size_t> rowrange,
	std::pair<size_t, size_t> colrange) const
{
	if (rowrange.second > A.M() - 1 || rowrange.first < 0 || colrange.second > A.N() - 1 || colrange.first < 0)
		throw std::invalid_argument("Invalid range parameters for taking a subset");
	if (rowrange.second < rowrange.first || colrange.second < colrange.first)
		throw std::invalid_argument("Invalid range arguments");

	size_t B_M = rowrange.second - rowrange.first + 1;
	size_t B_N = colrange.second - colrange.first + 1;
	
	cuda_gpu_matrix ans(B_M, B_N);

	double* src = A.c_ptr(), *dest = ans.c_ptr();


	// if the row range is the same, a single contiguous block
	// otherwise we need to copy in chunks

	if (B_M = A.M())
	{
		size_t start_dest = columnMajorZeroIndex(0, colrange.first, A.M(), A.N());
		checkCudaError<cuda_memory_error>(
			hipMemcpy(dest, src + start_dest, A.M() * B_N * sizeof(*(src)), hipMemcpyDeviceToDevice));
		return ans;
	}

	for (size_t start_col = 0; start_col < B_N; ++start_col)
	{
		double* start_dest = dest + columnMajorZeroIndex(rowrange.first, start_col + colrange.first, A.M(), A.N());
		double* start_src = src + columnMajorZeroIndex(0, start_col, ans.M(), ans.N());
		checkCudaError<cuda_memory_error>(
			hipMemcpy(start_dest, start_src, ans.M() * sizeof(*(src)), hipMemcpyDeviceToDevice));
	}
	return ans;
}

cuda_gpu_matrix 
linear_algebra::transpose(const cuda_gpu_matrix& A) const
{
	cuda_gpu_matrix ans = A.deep_copy();

	size_t block1Dim = 1 << 5;
	dim3 gridDim((A.M() + block1Dim - 1) / block1Dim, (A.N() + block1Dim - 1) / block1Dim);
	dim3 blockDim(block1Dim, block1Dim);

	transpose_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), ans.c_ptr(), A.M(), A.N());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return ans;
}

cuda_gpu_matrix
linear_algebra::find_column_maxes(const cuda_gpu_matrix& A) const
{
	cuda_gpu_matrix scaling(1, A.N());
	size_t blockDim = 1 << 5;
	size_t gridDim = (A.N() + blockDim - 1) / blockDim;

	find_column_maxes_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), scaling.c_ptr(), A.M(), A.N());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return scaling;
}

void 
linear_algebra::column_normalize(cuda_gpu_matrix& A, const cuda_gpu_matrix& scaling) const
{
	if (A.N() != scaling.N() || scaling.M() != 1)
		throw std::invalid_argument("incompatible dimensions for applying column normalization operations");


	size_t blockDim1 = 1 << 5;
	auto getGridDim = [blockDim1](size_t r) -> size_t { return (r + blockDim1 - 1) / blockDim1; };
	dim3 blockDim(blockDim1, blockDim1);
	dim3 gridDim = (getGridDim(A.M()), getGridDim(A.N()));
	column_normalize_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), scaling.c_ptr(), A.M(), A.N());
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
}

cuda_gpu_vector
linear_algebra::get_ones(size_t n) const
{
	cuda_gpu_vector ans(n);
	size_t blockDim = 1 << 5;
	size_t gridDim = (n + blockDim - 1) / blockDim;
	set_ones_kernel KERNEL_ARGS2(gridDim, blockDim) (ans.c_ptr(), n);
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return ans;
}

cuda_gpu_matrix 
linear_algebra::get_matrix_squared(const cuda_gpu_matrix& A) const
{
	size_t width = A.N() * (A.N() + 1) / 2;
	cuda_gpu_matrix ans(A.M(), width);

	std::shared_ptr<size_t> lookup_tbl(new size_t[A.N()], std::default_delete<size_t[]>());
	for (size_t i = 1; i <= A.N(); ++i)
	{
		lookup_tbl.get()[i - 1] = (i * i + i) / 2 - 1;
	}

	size_t block1Dim = 1 << 5;
	dim3 gridDim((A.M() + block1Dim - 1) / block1Dim, (width + block1Dim - 1) / block1Dim);
	dim3 blockDim(block1Dim, block1Dim);

	get_matrix_squared_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), A.M(), A.N(), ans.c_ptr(), lookup_tbl.get());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());

	return ans;
}

cuda_gpu_matrix
linear_algebra::tikhonov(const cuda_gpu_matrix& A, const cuda_gpu_matrix& b, double k) const
{
	cuda_gpu_matrix zeromat(A.N(), b.N());

	cuda_host_matrix identity_host(A.N(), A.N());
	for (size_t i = 0; i < A.N(); ++i)
	{
		identity_host[i][i] = sqrt(k);
	}
	cuda_gpu_matrix identity(A.N(), A.N());
	identity_host.copyToGpuMemory(identity);

	auto Aplus = concatenate(A, identity, true);
	auto bplus = concatenate(b, zeromat, true);

	auto pinv_Aplus = pinv(Aplus);
	auto solution = multiply(pinv_Aplus, false, bplus, false);
	return solution;
}

