
#include "hip/hip_runtime.h"
#include ""

#include "../include/cuda_host_matrix.h"
#include "../include/cuda_gpu_matrix.h"
#include "../include/cuda_libraries.h"
#include "../include/ddt.cuh"
#include "../include/gpu_timer.h"
#include "../include/linear_algebra.cuh"

#include <stdio.h>
#include <iostream>

int main()
{
    gpu_timer<true> block_timer("int main()");
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    using std::cout;
    using std::endl;

    cuda_libraries cuLibraries;
    linear_algebra linalg(cuLibraries);
    Ddt<ForwardDifference4> ddt(0.1);

    gpu_timer<false> timer("main timer");

    timer.start();
    cuda_host_matrix A(10, 10, cuda_host_matrix::MatrixType::CM_DENSE);
    for (int r = 0; r < 10 * 10; ++r)
        A.c_ptr()[r] = r/ 10.0;

    cuda_gpu_matrix Agpu(10, 10);
    A.copy_to_gpu_memory(Agpu);

    timer.stop();
    timer.elapsed();

    cuda_gpu_matrix ddt_ans = ddt(Agpu);

    timer.start();
    auto Cgpu = linalg.multiply(Agpu, false, Agpu, false);
    cuda_host_matrix C(Cgpu.M(), Cgpu.N(), cuda_host_matrix::MatrixType::CM_DENSE);
    timer.stop();
    timer.elapsed();

    C.copy_to_host_memory(Cgpu);

    A.print();
    C.print();
    return 0;
}
