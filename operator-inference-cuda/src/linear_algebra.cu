#include "hip/hip_runtime.h"
#include "../include/linear_algebra.cuh"

cuda_host_matrix create_host_matrix_from_gpu(cuda_gpu_matrix& A)
{
	cuda_host_matrix ans(A.M(), A.N());
	ans.copy_to_host_memory(A);
	return ans;
};

cuda_gpu_matrix create_gpu_matrix_from_host(cuda_host_matrix& A)
{
	cuda_gpu_matrix ans(A.M(), A.N());
	A.copy_to_gpu_memory(ans);
	return ans;
};

cuda_gpu_matrix
linear_algebra::multiply(
	const cuda_gpu_matrix& A,
	bool transposeA,
	const cuda_gpu_matrix& B,
	bool transposeB) const
{
	int A_M = transposeA ? A.N() : A.M();
	int A_N = transposeA ? A.M() : A.N();
	int B_M = transposeB ? B.N() : B.M();
	int B_N = transposeB ? B.M() : B.N();

	if (A_N != B_M)
		throw incompatible_dimensions_error(A_M, A_N, B_M, B_N);

	cuda_gpu_matrix C(A_M, B_N);

	// Configure matrix operation description
	hipblasOperation_t opAvalue = transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasOperation_t opBvalue = transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	double alpha = 1, beta = 0;

	hipblasDgemm(cudalibraries.get_blas_handle(),
		opAvalue, opBvalue,
		A_M, B_N, A_N,
		&alpha,
		A.c_ptr(),
		A.M(),
		B.c_ptr(),
		B.M(),
		&beta,
		C.c_ptr(),
		C.M());

	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return C;
};

cuda_gpu_vector
linear_algebra::add(
	const cuda_gpu_vector& a,
	double alpha,
	const cuda_gpu_vector& b,
	double beta)
{
	if (a.M() != b.M())
		throw incompatible_dimensions_error(a.M(), a.N(), b.M(), b.N());

	cuda_gpu_vector ans(a.M());
	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasDaxpy(cudalibraries.get_blas_handle(), a.M(), &alpha, a.c_ptr(), 1, ans.c_ptr(), 1));

	checkCudaStatus<cublas_matrix_operation_error>(
		hipblasDaxpy(cudalibraries.get_blas_handle(), a.M(), &beta, b.c_ptr(), 1, ans.c_ptr(), 1));

	return ans;
}


cuda_gpu_matrix
linear_algebra::concatenate(
		const cuda_gpu_matrix& A,
		const cuda_gpu_matrix& B,
		bool shouldConcatenateVertically) const
{
	if (shouldConcatenateVertically && A.N() != B.N() || !shouldConcatenateVertically && A.M() != B.M())
		throw std::invalid_argument("Invalid dimensions for specified concatenation operation");
	
	double* Asrc = A.c_ptr();
	double* Bsrc = B.c_ptr();
	if (!shouldConcatenateVertically)
	{
		cuda_gpu_matrix ans(A.M(), A.N() + B.N());
		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr(),
			Asrc,
			A.M() * A.N() * sizeof((*Asrc)),
			hipMemcpyDeviceToDevice));

		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + A.M() * A.N(),
			Bsrc,
			B.M() * B.N() * sizeof((*Bsrc)),
			hipMemcpyDeviceToDevice));
		return ans;
	}

	cuda_gpu_matrix ans(A.M() + B.M(), A.N());
	for (size_t col = 0; col < A.N(); ++col)
	{
		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + columnMajorZeroIndex(0, col, ans.M(), ans.N()),
			Asrc + columnMajorZeroIndex(0, col, A.M(), A.N()),
			A.M() * sizeof((*Asrc)),
			hipMemcpyDeviceToDevice));

		checkCudaError<cuda_memory_error>(hipMemcpy(
			ans.c_ptr() + columnMajorZeroIndex(A.M(), col, ans.M(), ans.N()),
			Bsrc + columnMajorZeroIndex(0, col, B.M(), B.N()),
			B.M() * sizeof((*Bsrc)),
			hipMemcpyDeviceToDevice));
	}
	return ans;
}

svd
linear_algebra::svd_decomposition(const cuda_gpu_matrix& A) const
{
	int lwork;
	checkCudaStatus<cuda_memory_error>(
		hipsolverDnSgesvd_bufferSize(this->cudalibraries.get_solver_handle(), A.M(), A.N(), &lwork));

	std::shared_ptr<double> workspace = allocate_on_device<double>(sizeof(double) * lwork);

	signed char jobu = 'A', jobvt = 'A';

	size_t diag_length = A.M() < A.N() ? A.M() : A.N();

	cuda_gpu_matrix U(A.M(), A.M());
	cuda_gpu_vector Svec(diag_length);
	cuda_gpu_vector unconverged(diag_length);
	cuda_gpu_matrix Vt(A.N(), A.N());

	double* d_work, *d_rwork;
	int* dev_info = 0;

	hipsolverDnDgesvd(
		this->cudalibraries.get_solver_handle(),
		jobu,
		jobvt,
		A.M(),
		A.N(),
		A.c_ptr(),
		A.M(),
		Svec.c_ptr(),
		U.c_ptr(),
		U.M(),  // ldu
		Vt.c_ptr(),
		Vt.M(), // ldvt,
		workspace.get(),
		lwork,
		unconverged.c_ptr(),
		dev_info);

	cuda_gpu_matrix S(A.M(), A.N());
	for (size_t i = 0; i < diag_length; ++i)
	{
		hipMemcpy(
			S.c_ptr() + columnMajorZeroIndex(i, i, A.M(), A.N()),
			Svec.c_ptr() + columnMajorZeroIndex(i, 1, Svec.M(), Svec.N()),
			sizeof(double),
			hipMemcpyDeviceToDevice);
	}
	svd ans{ U, S, Vt };
	return ans;
}

cuda_gpu_matrix
linear_algebra::pinv(const cuda_gpu_matrix& A) const
{
	return pinv(svd_decomposition(A));
}

cuda_gpu_matrix
linear_algebra::pinv(const svd& decomposition) const
{
	cuda_gpu_matrix pinvSigma = decomposition.S.deep_copy();

	// invert diagonal matrix kernel
	size_t diag_length = pinvSigma.M() > pinvSigma.N() ? pinvSigma.N() : pinvSigma.M();

	size_t blockDim = 1 << 6;
	size_t gridDim = (diag_length + blockDim - 1) / blockDim;
	
	invert_rectangular_diagonal_kernel KERNEL_ARGS2(gridDim, blockDim) (pinvSigma.c_ptr(), pinvSigma.M(), pinvSigma.N(), std::numeric_limits<double>::epsilon());
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());

	cuda_gpu_matrix pinv = multiply(
		multiply(
			decomposition.Vt, true,
			pinvSigma, true), false, 
		decomposition.U, true);

	return pinv;
}

cuda_gpu_matrix
linear_algebra::subset(
	const cuda_gpu_matrix& A,
	std::pair<size_t, size_t> rowrange,
	std::pair<size_t, size_t> colrange) const
{
	if (rowrange.second > A.M() - 1 || rowrange.first < 0 || colrange.second > A.N() - 1 || colrange.first < 0)
		throw std::invalid_argument("Invalid range parameters for taking a subset");
	if (rowrange.second < rowrange.first || colrange.second < colrange.first)
		throw std::invalid_argument("Invalid range arguments");

	size_t B_M = rowrange.second - rowrange.first + 1;
	size_t B_N = colrange.second - colrange.first + 1;
	
	cuda_gpu_matrix ans(B_M, B_N);

	// if the row range is the same, a single contiguous block
	// otherwise we need to copy in chunks

	if (B_M == A.M())
	{
		size_t start_dest = columnMajorZeroIndex(0, colrange.first, A.M(), A.N());
		checkCudaError<cuda_memory_error>(
			hipMemcpy(ans.c_ptr(), A.c_ptr() + start_dest, A.M() * B_N * sizeof(*(A.c_ptr())), hipMemcpyDeviceToDevice));
		return ans;
	}

	for (size_t start_col = 0; start_col < B_N; ++start_col)
	{
		double* start_src = A.c_ptr() + columnMajorZeroIndex(rowrange.first, start_col + colrange.first, A.M(), A.N());
		double* start_dest = ans.c_ptr() + columnMajorZeroIndex(0, start_col, ans.M(), ans.N());
		checkCudaError<cuda_memory_error>(
			hipMemcpy(start_dest, start_src, ans.M() * sizeof(double), hipMemcpyDeviceToDevice));
	}
	return ans;
}

cuda_gpu_matrix 
linear_algebra::transpose(const cuda_gpu_matrix& A) const
{
	cuda_gpu_matrix ans = A.deep_copy();
	using std::swap;
	swap(ans._m, ans._n);

	size_t block1Dim = 1 << 5;
	dim3 gridDim((A.M() + block1Dim - 1) / block1Dim, (A.N() + block1Dim - 1) / block1Dim);
	dim3 blockDim(block1Dim, block1Dim);

	transpose_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), ans.c_ptr(), A.M(), A.N());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return ans;
}

cuda_gpu_matrix
linear_algebra::find_column_abs_maxes(const cuda_gpu_matrix& A) const
{
	cuda_gpu_matrix scaling(1, A.N());
	size_t blockDim = 1 << 5;
	size_t gridDim = (A.N() + blockDim - 1) / blockDim;

	find_column_abs_maxes_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), scaling.c_ptr(), A.M(), A.N());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return scaling;
}

void 
linear_algebra::column_normalize(cuda_gpu_matrix& A, const cuda_gpu_matrix& scaling) const
{
	if (A.N() != scaling.N() || scaling.M() != 1)
		throw std::invalid_argument("incompatible dimensions for applying column normalization operations");


	size_t blockDim1 = 1 << 5;
	auto getGridDim = [blockDim1](size_t r) -> size_t { return (r + blockDim1 - 1) / blockDim1; };
	dim3 blockDim(blockDim1, blockDim1);
	dim3 gridDim(getGridDim(A.M()), getGridDim(A.N()));

	column_normalize_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), scaling.c_ptr(), A.M(), A.N());
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
}

cuda_gpu_vector
linear_algebra::get_ones(size_t n) const
{
	cuda_gpu_vector ans(n);
	size_t blockDim = 1 << 5;
	size_t gridDim = (n + blockDim - 1) / blockDim;
	set_ones_kernel KERNEL_ARGS2(gridDim, blockDim) (ans.c_ptr(), n);
	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());
	return ans;
}

cuda_gpu_matrix 
linear_algebra::get_matrix_squared(const cuda_gpu_matrix& A) const
{
	size_t width = A.N() * (A.N() + 1) / 2;
	cuda_gpu_matrix ans(A.M(), width);
	std::shared_ptr<size_t> lookup_table = get_lookup_table(A.N());

	size_t block1Dim = 1 << 5;
	dim3 gridDim((A.M() + block1Dim - 1) / block1Dim, (width + block1Dim - 1) / block1Dim);
	dim3 blockDim(block1Dim, block1Dim);

	get_matrix_squared_kernel KERNEL_ARGS2(gridDim, blockDim) (A.c_ptr(), A.M(), A.N(), ans.c_ptr(), lookup_table.get());

	checkCudaError<cublas_matrix_operation_error>(hipGetLastError());
	checkCudaError<cublas_matrix_operation_error>(hipDeviceSynchronize());

	return ans;
}

std::shared_ptr<size_t>
linear_algebra::get_lookup_table(size_t N) const
{
	std::unique_ptr<size_t[]> host_lookup = std::make_unique<size_t[]>(N);
	std::shared_ptr<size_t> device_lookup = allocate_on_device<size_t>(sizeof(size_t) * N);

	size_t* host_ptr = host_lookup.get();
	host_ptr[0] = N - 1;
	for (int i = N - 2; i >= 0; --i)
	{
		host_ptr[N - i - 1] = (size_t)(i + 1) + host_ptr[N - i - 2];
	}

	checkCudaError<cuda_memory_error>(
		hipMemcpy(
			device_lookup.get(),
			host_lookup.get(),
			sizeof(size_t) * N,
			hipMemcpyHostToDevice));

	return device_lookup;
}


cuda_gpu_matrix
linear_algebra::tikhonov(const cuda_gpu_matrix& A, const cuda_gpu_matrix& b, double k) const
{
	cuda_gpu_matrix zeromat(A.N(), b.N());

	cuda_host_matrix identity_host(A.N(), A.N());
	for (size_t i = 0; i < A.N(); ++i)
	{
		identity_host[i][i] = sqrt(k);
	}

	cuda_gpu_matrix identity = create_gpu_matrix_from_host(identity_host);
	auto Aplus = concatenate(A, identity, true);
	auto bplus = concatenate(b, zeromat, true);

	auto pinv_Aplus = pinv(Aplus);
	auto solution = multiply(pinv_Aplus, false, bplus, false);
	return solution;
}

